#include "cuda/genetic_operation.cuh"

__device__ 
Individual device_random_all_constants(Individual ind, hiprandState *state){
	for(size_t i = 0; i != ind.size; i++){
		if(ind.node[i].type == TERMINAL){
			if(ind.node[i].flag == CONST){
				ind.node[i].value.constant = hiprand_uniform(state)*100;
			}
		}
	}
	return ind;
}


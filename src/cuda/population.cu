#include "hip/hip_runtime.h"
#include "cuda/population.cuh"

Population* cuda_malloc_population(){
	Population* d_pop;
	if(hipMalloc(&d_pop, sizeof(Population)) != hipSuccess){
		// TODO Implement error handlig
		printf("Cuda device allocation fail ");
	};
	return d_pop;
}

void cuda_host_to_device_memcpy_population(Population* d_pop, Population* pop){
	if(hipMemcpy(d_pop, pop, sizeof(Population), hipMemcpyHostToDevice) != hipSuccess){
		// TODO Implement error handlig
		printf("Cuda host to device copy fail ");
	}
}
void cuda_device_to_host_memcpy_population(Population* d_pop, Population* pop){
	if(hipMemcpy(pop, d_pop, sizeof(Population), hipMemcpyDeviceToHost) != hipSuccess){
		// TODO Implement error handlig
		printf("Cuda device to host copy fail ");
	}

}

__global__
void device_get_n_indexes_of_best_ind_in_population(const Population* pop, const size_t n, size_t* indexes){
	Population* new_pop = (Population*)(malloc(sizeof(Population)));
	*new_pop = *pop;
	for(size_t a = 0; a != n; a++){
		float best = FLT_MAX;
		size_t best_index = 0;
		for(size_t i = 0; i != pop->size; ++i){
			if(new_pop->ind[i].fitness < best){
				best = new_pop->ind[i].fitness;
				best_index = i;
			}
		}
		indexes[a] = best_index;
		new_pop->ind[best_index].fitness = FLT_MAX;
	}
	free(new_pop);
};	

__global__
void device_print_best_so_far(const Population* d_pop){
	Individual best;
	best.fitness = FLT_MAX;
	for(size_t i = 0; i != d_pop->size; ++i){
		if(d_pop->ind[i].fitness <= best.fitness){
			best = d_pop->ind[i];
		}
	}
	printf("Best so far:\n");
	device_print_ind(&best);
	printf("with fitness: %f.\n", best.fitness);
}

__global__
void device_check_end_condition(const Population* d_pop){
	Individual best;
	best.fitness = FLT_MAX;
	for(size_t i = 0; i != d_pop->size; ++i){
		if(d_pop->ind[i].fitness <= best.fitness){
			best = d_pop->ind[i];
		}
	}
	if(best.fitness <= 0.1){
		printf("############################\n");
		printf("Found solution: \n");
		device_print_ind(&best);
		printf("with fitness: %f.\n", best.fitness);
	}
}

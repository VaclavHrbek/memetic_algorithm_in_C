#include "hip/hip_runtime.h"
#include "cuda/population.cuh"

Population* cuda_malloc_population(){
	Population* d_pop;
	if(hipMalloc(&d_pop, sizeof(Population)) != hipSuccess){
		hipError_t err = hipGetLastError();
		printf("Error in function: %s\n"
				"Error name: %s \n"
				"Error description: %s\n",
				__FUNCTION__,
				hipGetErrorName(err),
				hipGetErrorString(err));
	};
	return d_pop;
}

void cuda_host_to_device_memcpy_population(Population* d_pop, Population* pop){
	if(hipMemcpy(d_pop, pop, sizeof(Population), hipMemcpyHostToDevice) != hipSuccess){
		hipError_t err = hipGetLastError();
		printf("Error in function: %s\n"
				"Error name: %s \n"
				"Error description: %s\n",
				__FUNCTION__,
				hipGetErrorName(err),
				hipGetErrorString(err));
	}
}
void cuda_device_to_host_memcpy_population(Population* d_pop, Population* pop){
	if(hipMemcpy(pop, d_pop, sizeof(Population), hipMemcpyDeviceToHost) != hipSuccess){
		hipError_t err = hipGetLastError();
		printf("Error in function: %s\n"
				"Error name: %s \n"
				"Error description: %s\n",
				__FUNCTION__,
				hipGetErrorName(err),
				hipGetErrorString(err));
	}

}

__global__
void device_get_n_indexes_of_best_ind_in_population(const Population* pop, const size_t n, size_t* indexes){
	Population* new_pop = (Population*)(malloc(sizeof(Population)));
	*new_pop = *pop;
	for(size_t a = 0; a != n; a++){
		float best = FLT_MAX;
		size_t best_index = 0;
		for(size_t i = 0; i != pop->size; ++i){
			if(new_pop->ind[i].fitness < best){
				best = new_pop->ind[i].fitness;
				best_index = i;
			}
		}
		indexes[a] = best_index;
		new_pop->ind[best_index].fitness = FLT_MAX;
	}
	free(new_pop);
};	

__global__
void device_print_best_so_far(const Population* d_pop){
	Individual best;
	best.fitness = FLT_MAX;
	for(size_t i = 0; i != d_pop->size; ++i){
		if(d_pop->ind[i].fitness <= best.fitness){
			best = d_pop->ind[i];
		}
	}
	printf("Best so far,");
	printf(" %f,", best.fitness);
	device_print_ind(&best);
}

__global__
void device_check_end_condition(const Population* d_pop, int *flag){
	Individual best;
	best.fitness = FLT_MAX;
	for(size_t i = 0; i != d_pop->size; ++i){
		if(d_pop->ind[i].fitness <= best.fitness){
			best = d_pop->ind[i];
		}
	}
	if(*flag == 0){
	if(best.fitness <= 1){
			*flag = 1;
			printf("Found solution, ");
			printf("%f,", best.fitness);
			device_print_ind(&best);
		}
	}
}

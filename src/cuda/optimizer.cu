#include "hip/hip_runtime.h"
#include "cuda/optimizer.cuh"

__device__
Individual device_hill_climbing(Individual ind, hiprandState* state);
__global__
void device_optimize(Population* d_pop, hiprandState* state, size_t* indexes);
__device__
Individual device_random_search(Individual ind, hiprandState* state);
__global__ 
void setup_kernel(hiprandState *state);

void cuda_optimize(Population* d_pop){
	int total_threads = 256;
	dim3 block(total_threads);
	dim3 grid((SIZE_FOR_OPTIMIZATION / block.x) + 1); 
	size_t* d_n_best_indexes_in_pop;
	hipMalloc(&d_n_best_indexes_in_pop, sizeof(size_t)*SIZE_FOR_OPTIMIZATION);
	hiprandState *dev_state;
	hipMalloc((void**)&dev_state, (block.x * grid.x) * sizeof(hiprandState));
	setup_kernel<<<grid, block>>>(dev_state);
	device_get_n_indexes_of_best_ind_in_population<<<1,1>>>(d_pop, SIZE_FOR_OPTIMIZATION, d_n_best_indexes_in_pop);
	device_optimize<<<grid, block>>>(d_pop, dev_state, d_n_best_indexes_in_pop);
	hipDeviceSynchronize();
}

__global__ void setup_kernel(hiprandState *state)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets same seed, a different sequence
       number, no offset */
	if(i < SIZE_FOR_OPTIMIZATION){
		hiprand_init(1234, i, 0, &state[i]);
	}
}

__global__
void device_optimize(Population* d_pop, hiprandState *state, size_t* indexes){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i < SIZE_FOR_OPTIMIZATION){
		/* Copy state to local memory for efficiency */
		hiprandState local_state = state[i];
		Individual* ind = &d_pop->ind[indexes[i]];
		for(size_t i = 0; i != NUM_OF_OPTIMIZATION; ++i){
			//Individual ind_2 = device_hill_climbing(*ind, &local_state);
			Individual ind_2 = device_random_search(*ind, &local_state);
			ind_2.fitness = device_equation(ind_2);
			if(ind_2.fitness < ind->fitness){
				*ind = ind_2;
				//printf("Better solution found in optimizer \n");
			}
		}
	}
}

__device__
Individual device_random_search(Individual ind, hiprandState* state){
	return device_random_all_constants(ind, state);
}

__device__
Individual device_hill_climbing(Individual ind, hiprandState* state){
	for(size_t i = 0; i != ind.size; i++){
		if(ind.node[i].type == TERMINAL){
			if(ind.node[i].flag == CONST){
				double curr_fit = 0;
				do{
					curr_fit = device_equation(ind);
					ind.node[i].value.constant += 1;
				}while(device_equation(ind) < curr_fit);
				ind.node[i].value.constant -= 1;
				do{
					curr_fit = device_equation(ind);
					ind.node[i].value.constant -= 1;
				}while(device_equation(ind) < curr_fit);
				ind.node[i].value.constant += 1;
			}
		}
	}
	return ind;
}

#include "hip/hip_runtime.h"
#include "cuda/optimizer.cuh"

__global__
void device_optimize(Population* d_pop, hiprandState* state, size_t* indexes);
__device__
Individual device_stochastic_hill_climbing_constants(Individual ind, hiprandState* state);
__global__ 
void setup_kernel(hiprandState *state);

void cuda_optimize(Population* d_pop){
	int total_threads = 256;
	dim3 block(total_threads);
	dim3 grid((SIZE_FOR_OPTIMIZATION / block.x) + 1); 
	size_t* d_n_best_indexes_in_pop;
	hipMalloc(&d_n_best_indexes_in_pop, sizeof(size_t)*SIZE_FOR_OPTIMIZATION);
	hiprandState *dev_state;
	hipMalloc((void**)&dev_state, (block.x * grid.x) * sizeof(hiprandState));

	setup_kernel<<<grid, block>>>(dev_state);
	device_get_n_indexes_of_best_ind_in_population<<<1,1>>>(d_pop, SIZE_FOR_OPTIMIZATION, d_n_best_indexes_in_pop);
	device_optimize<<<grid, block>>>(d_pop, dev_state, d_n_best_indexes_in_pop);
}

__global__ void setup_kernel(hiprandState *state)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets same seed, a different sequence
       number, no offset */
	if(i < SIZE_FOR_OPTIMIZATION){
		hiprand_init(1234, i, 0, &state[i]);
	}
}

__global__
void device_optimize(Population* d_pop, hiprandState *state, size_t* indexes){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i < SIZE_FOR_OPTIMIZATION){
		/* Copy state to local memory for efficiency */
		hiprandState local_state = state[i];
		Individual* ind = &d_pop->ind[indexes[i]];
		for(size_t i = 0; i != NUM_OF_OPTIMIZATION; ++i){
			Individual ind_2 = device_stochastic_hill_climbing_constants(*ind, &local_state);
			ind_2.fitness = device_equation(ind_2);
			if(ind_2.fitness < ind->fitness){
				*ind = ind_2;
				printf("Better solution found in optimizer \n");
			}
		}
	}
}

__device__
Individual device_stochastic_hill_climbing_constants(Individual ind, hiprandState* state){
	Individual ind_2 = device_random_all_constants(ind, state);
	return ind;
}

#include "hip/hip_runtime.h"
#include "cuda/genetic_program.cuh"

void run_gp_gpu(GeneticProgram* gp){
	srand((unsigned int)time(NULL));	
	clock_t start_program = clock();
	clock_t duration_cycle = 0;

	Population* d_pop = cuda_malloc_population();

	for(size_t a = 0; a != NUM_OF_EVOLUTIONARY_CYCLES; ++a){
		gp->pop = malloc_population();
		cuda_host_to_device_memcpy_population(d_pop, gp->pop);
		printf("#####################\n"
				"Evolutionray cycle %li\n", a);
		clock_t start_evo_cycle = clock();
		clock_t duration_optimize = 0;
		clock_t duration_of_gen = 0;
		clock_t duration_gen_operation = 0;
		clock_t duration_cal_fit = 0;

		cuda_calculate_fitness(d_pop);
		for(size_t i = 0; i != NUM_OF_GENERATION; ++i){
			printf("Generation: %ld \n", i);
			clock_t start_generation = clock();

			cuda_calculate_fitness(d_pop);
			//TODO use for_each algorithm on next function (if it is possible
			// because I don't use the containers).
			device_check_end_condition<<<1,1>>>(d_pop);
			device_print_best_so_far<<<1,1>>>(d_pop);

			clock_t start_optimize = clock();
			cuda_optimize(d_pop);
			duration_optimize += clock() - start_optimize;

			clock_t start_cal_fit = clock();
			cuda_calculate_fitness(d_pop);
			duration_cal_fit += clock() - start_cal_fit;
			device_check_end_condition<<<1,1>>>(d_pop);

			clock_t start_gen_operation = clock();
			cuda_device_to_host_memcpy_population(d_pop, gp->pop);	
			gp->pop = genetic_operations(gp->pop, MUTAION_RATE, CROSSOUVER_RATE);
			cuda_host_to_device_memcpy_population(d_pop, gp->pop);
			duration_gen_operation += clock() - start_gen_operation;

			duration_of_gen += clock() - start_generation;
		}
		printf("####################\n\n \
				Mean duration of optimization: %f s\n \
				Mean duration of generation: %f s\n \
				Mean duration of calculation fitness: %f s\n \
				Mean duration of genetic operation + plus transfer from device and to device: %f s\n \
				", ((double)duration_optimize/CLOCKS_PER_SEC)/NUM_OF_GENERATION
				, ((double)duration_of_gen/CLOCKS_PER_SEC)/NUM_OF_GENERATION
				, ((double)duration_cal_fit/CLOCKS_PER_SEC)/NUM_OF_GENERATION
				, ((double)duration_gen_operation/CLOCKS_PER_SEC)/NUM_OF_GENERATION);

		duration_cycle += clock() - start_evo_cycle;

	}
	printf("####################\n\n \
			Mean duration of evolutionary cycle: %f s\n \
			\n ######################", ((double)duration_cycle/CLOCKS_PER_SEC)/NUM_OF_EVOLUTIONARY_CYCLES);

	clock_t duration_program = clock() - start_program;
	printf("####################\n\n \
			Duration of program %f s\n \
			\n ######################", (double)duration_program/CLOCKS_PER_SEC);
	print_best_ind_from_population(gp->pop);
};

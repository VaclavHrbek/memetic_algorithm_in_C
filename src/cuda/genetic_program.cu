#include "hip/hip_runtime.h"
#include "cuda/genetic_program.cuh"

void run_gp_gpu(GeneticProgram* gp){
	srand((unsigned int)time(NULL));	
	clock_t start_program = clock();
	clock_t duration_cycle = 0;
	std::ofstream out_file;
	out_file.open(OUTPUT_FILE_NAME, std::ios_base::app);
	out_file << "Mean duration of generation," <<
	"Mean duration of optimization," <<
	"Mean duration of calculation fitness," <<
	"Mean duration of genetic operation," <<
	"Number of generation required to found solution\n";
	out_file.close();

	Population* d_pop = cuda_malloc_population();

	for(size_t a = 0; a != NUM_OF_EVOLUTIONARY_CYCLES; ++a){
		gp->pop = malloc_population();
		int *flag;
		hipMallocManaged(&flag, sizeof(int));
		flag[0] = 0;
		cuda_host_to_device_memcpy_population(d_pop, gp->pop);
		printf("#####################\n"
				"Evolutionray cycle %li\n", a);
		clock_t start_evo_cycle = clock();
		clock_t duration_optimize = 0;
		clock_t duration_of_gen = 0;
		clock_t duration_gen_operation = 0;
		clock_t duration_cal_fit = 0;

		cuda_calculate_fitness(d_pop);
		size_t i = 0;
		for(i = 0; i != NUM_OF_GENERATION; ++i){
			//printf("Generation: %ld \n", i);
			clock_t start_generation = clock();

			cuda_calculate_fitness(d_pop);
			//TODO use for_each algorithm on next function (if it is possible
			// because I don't use the containers).
			device_check_end_condition<<<1,1>>>(d_pop, flag);
			hipDeviceSynchronize();
			if(flag[0] == 1){
				break;
			}
			//device_print_best_so_far<<<1,1>>>(d_pop);

			clock_t start_optimize = clock();
			cuda_optimize(d_pop);
			duration_optimize += clock() - start_optimize;

			clock_t start_cal_fit = clock();
			cuda_calculate_fitness(d_pop);
			duration_cal_fit += clock() - start_cal_fit;

			device_check_end_condition<<<1,1>>>(d_pop, flag);
			hipDeviceSynchronize();
			if(flag[0] == 1){
				break;
			}

			clock_t start_gen_operation = clock();
			cuda_device_to_host_memcpy_population(d_pop, gp->pop);	
			gp->pop = genetic_operations(gp->pop, MUTAION_RATE, CROSSOUVER_RATE);
			if(check_fitness_less_than(gp->pop, END_CONDITION_FITTNESS_LESS_THAN) == true){
				std::ofstream out_run_file;
				out_run_file.open(OUTPUT_RUN_FILE, std::ios_base::app);
				out_run_file << "Found solution after genetic operation,";
				out_run_file.close();
				put_best_to_file(gp->pop);
				break;
			};
			cuda_host_to_device_memcpy_population(d_pop, gp->pop);
			duration_gen_operation += clock() - start_gen_operation;

			duration_of_gen += clock() - start_generation;
		}
		std::ofstream out_file;
		out_file.open(OUTPUT_FILE_NAME, std::ios_base::app);
		out_file << ((double)duration_of_gen/CLOCKS_PER_SEC)/i << ',' <<
			 ((double)duration_optimize/CLOCKS_PER_SEC)/i << ',' <<
			 ((double)duration_cal_fit/CLOCKS_PER_SEC)/i<< ',' <<
			 ((double)duration_gen_operation/CLOCKS_PER_SEC)/i << 
			 ',' << i << '\n';
		duration_cycle += clock() - start_evo_cycle;
		out_file.close();
		duration_cycle += clock() - start_evo_cycle;
	}
	clock_t duration_program = clock() - start_program;
	out_file.open(OUTPUT_FILE_NAME, std::ios_base::app);
	out_file << "Mean duration of evolutionary cycle," << ((double)duration_cycle/CLOCKS_PER_SEC)/NUM_OF_EVOLUTIONARY_CYCLES << '\n';
	out_file << "Duration of program," << ((double)duration_program/CLOCKS_PER_SEC) << '\n';
	out_file.close();
};

#include "hip/hip_runtime.h"
#include "cuda/fitness.cuh"

__global__
void device_calculate_fitness(Population* pop);

void cuda_calculate_fitness(Population* pop){
	Population* d_pop;
	if(hipMalloc(&d_pop, sizeof(Population)) != hipSuccess){
		// Implement error handlig
		return;
	};
	hipMemcpy(d_pop, pop, sizeof(Population), hipMemcpyHostToDevice);

	dim3 block(1023);
	dim3 grid((pop->size_pop / block.x) + 1); 
	device_calculate_fitness<<<grid, block>>>(d_pop);

	hipMemcpy(pop, d_pop, sizeof(Population), hipMemcpyDeviceToHost);
	hipFree(d_pop);
}

__global__
void device_calculate_fitness(Population* pop){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i < pop->size_pop){
		pop->fitness[i] = device_equation(pop->arr[i], pop->num_of_nodes_ind);
	}
}	

#include "hip/hip_runtime.h"
#include "cuda/fitness.cuh"

__global__
void device_calculate_fitness(Population* pop);

void cuda_calculate_fitness(Population* pop){
	Population* d_pop;
	hipMalloc(&d_pop, sizeof(Population));
	hipMemcpy(d_pop, pop, sizeof(Population), hipMemcpyHostToDevice);
	device_calculate_fitness<<<1, SIZE_OF_POPULAION>>>(d_pop);
	hipMemcpy(pop, d_pop, sizeof(Population), hipMemcpyDeviceToHost);
	hipFree(d_pop);
}

__global__
void device_calculate_fitness(Population* pop){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	pop->fitness[i] = device_equation(pop->arr[i], pop->num_of_nodes_ind);
}	

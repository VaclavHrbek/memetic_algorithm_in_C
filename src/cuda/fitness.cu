#include "hip/hip_runtime.h"
#include "cuda/fitness.cuh"

__global__
void device_calculate_fitness(Population* pop);

void cuda_calculate_fitness(Population* d_pop){
	dim3 block(256);
	dim3 grid((SIZE_OF_POPULAION / block.x) + 1); 
	device_calculate_fitness<<<grid, block>>>(d_pop);
	hipDeviceSynchronize();
}

__global__
void device_calculate_fitness(Population* pop){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i < pop->size){
		pop->ind[i].fitness = device_equation_xy(pop->ind[i]); 
		if(pop->ind[i].fitness <= END_CONDITION_FITTNESS_LESS_THAN){
			printf("Found best solution with fitness: %f.\nOn index %i in population.\n", pop->ind[i].fitness, i);
			device_print_ind(&pop->ind[i]);
		}
	}
}	
